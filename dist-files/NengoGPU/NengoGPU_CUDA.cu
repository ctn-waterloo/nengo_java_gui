#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C"{
#endif

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "NengoGPU.h"
#include "NengoGPU_CUDA.h"

# define MAX_SHARED_MEM_SIZE 16000

// print the contents of an array of integers located on the device
void printIntArrayFromDevice(FILE* fp, intArray* a, int n, int m, int labels)
{
  int* temp = (int*) malloc( m * n * sizeof(int));
  hipMemcpy(temp, a->array, m * n * sizeof(int), hipMemcpyDeviceToHost);

  printf("%s:\n", a->name);

  int i, j;
  for(i = 0; i < n; i++)
  {
    fp ? fprintf(fp, "line %d: ", i) : printf("line %d:", i);
    for(j = 0; j < m; j++)
    {
      if(labels)
        fp ? fprintf(fp, "(%d, %d) ", j, temp[i * n + j]) : printf("(%d, %d) ", j, temp[i * n + j]);
      else
        fp ? fprintf(fp, "%d ", temp[i * n + j]) : printf("%d ", temp[i * n + j]);
    }
    fp ? fprintf(fp, "\n") : printf("\n");
  }

  fp ? fprintf(fp, "\n") : printf("\n");

  free(temp);
}

// print the contents of an array of floats located on the device
void printFloatArrayFromDevice(FILE* fp, floatArray* a, int n, int m, int labels)
{
  hipError_t err;
  float* temp = (float*) malloc( m * n * sizeof(float));
  err = hipMemcpy(temp, a->array, m * n * sizeof(float), hipMemcpyDeviceToHost);
  checkCudaError(err, "in printFloatArrayFromDevice, copying from device to host");

  printf("%s:\n", a->name);

  int i, j;
  for(i = 0; i < n; i++)
  {
    fp ? fprintf(fp, "line %d: ", i) : printf("line %d:", i);
    for(j = 0; j < m; j++)
    {
      if(labels)
        fp ? fprintf(fp, "(%d, %f) ", j, temp[i * n + j]) : printf("(%d, %f) ", j, temp[i * n + j]);
      else
        fp ? fprintf(fp, "%f ", temp[i * n + j]) : printf("%f ", temp[i * n + j]);
    }

    fp ? fprintf(fp, "\n") : printf("\n");
  }

  fp ? fprintf(fp, "\n") : printf("\n");

  free(temp);
}

void printIntColumn(FILE* fp, int* array, int m, int n, int col)
{
  int* temp = (int*) malloc( m * n * sizeof(int));
  hipMemcpy(temp, array, m * n * sizeof(int), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "%d ", temp[i * n + col]) : printf("%d ", temp[i * n + col]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

void printFloatColumn(FILE* fp, float* array, int m, int n, int col)
{
  float* temp = (float*) malloc( m * n * sizeof(float));
  hipMemcpy(temp, array, m * n * sizeof(float), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < m; i++)
  {
    fp ? fprintf(fp, "%f ", temp[i * n + col]) : printf("%f ", temp[i * n + col]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}
 
void printFloatRange(FILE* fp, float* array, int start, int end)
{
  float* temp = (float*) malloc((end - start + 1)  * sizeof(float));
  hipMemcpy(temp, array + start, (end - start + 1) * sizeof(float), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < end - start + 1; i++)
  {
    fp ? fprintf(fp, "%f ", temp[i]) : printf("%f ", temp[i]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

void printIntRange(FILE* fp, int* array, int start, int end)
{
  int* temp = (int*) malloc((end - start + 1)  * sizeof(int));
  hipMemcpy(temp, array + start, (end - start + 1) * sizeof(int), hipMemcpyDeviceToHost);

  int i;
  for(i = 0; i < end - start + 1; i++)
  {
    fp ? fprintf(fp, "%d ", temp[i]) : printf("%d ", temp[i]);
  }
  fp ? fprintf(fp, "\n") : printf("\n");
}

// get number of devices available
int getGPUDeviceCount(){
  hipError_t err;
  int numDevices;
  
  err = hipGetDeviceCount(&numDevices);
  checkCudaError(err, "get GPU device count");
  
  return numDevices;
}

// Reserves device with number deviceNum for the thread that calls this function. No interaction with the device should take place until this has been called.
// Once the device is reserved for the thread, no other thread should try to interact with that device or reserve it. A thread can reserve only one device at a time
void initGPUDevice(int deviceNum)
{
  hipSetDevice(deviceNum);
}

void shutdownGPUDevice()
{
}

void checkCudaErrorWithDevice(hipError_t err, int device, char* message)
{
  if(!err)
      return;

  printf("device: %d", device);
  checkCudaError(err, message);
}

void checkCudaError(hipError_t err, char* message)
{
    if(!err)
        return;

    printf(" CUDA ERROR: message: %s, description: %s\n", message, hipGetErrorString(err));

    exit(EXIT_FAILURE);
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalNumTerminationRows kernels are launched.
// Dot product the ith termination row with the corresponding input vector. Integrate the result. Results are stored in terminationValues. 
__global__ void transform(float dt, int numTransformRows, float* input, int* terminationOffsetInInput, int* transformRowToInputIndexor, float* transforms, float* tau, float* terminationOutput, int* terminationOutputIndexor, int* inputDimensions)
{
  
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < numTransformRows)
  {
    
    int j;
    int inputIndex = transformRowToInputIndexor[i];
    int offset = terminationOffsetInInput[inputIndex];
    
    int inputDimension = inputDimensions[inputIndex];
    int transformRowIndex = i;
    
    float my_tau = tau[inputIndex];
    
    float dot_product = 0;
    
    for(j=0; j < inputDimension; j++)
    {
      dot_product += input[offset + j] * transforms[transformRowIndex];

      transformRowIndex += numTransformRows;
    }
   
    float dt_over_tau = dt / my_tau;
    

    int outputIndex = terminationOutputIndexor[i];
    terminationOutput[outputIndex] = (1 - dt_over_tau) * terminationOutput[outputIndex] + dt_over_tau * dot_product;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalDimension kernels are launched.
// Sum the termination values for one dimension of one ensemble. Results are stored in ensembleSums.
__global__ void sumTerminations(int totalDimensions, int maxNumDecodedTerminations, float* terminationOutput, float* ensembleSums)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if( i < totalDimensions)
  {
    int terminationOutputIndex = i;
    int j;
    float sum = 0;

    for(j=0; j < maxNumDecodedTerminations; j++)
    {
      sum += terminationOutput[terminationOutputIndex];
      terminationOutputIndex += totalDimensions;
    }

    ensembleSums[i] = sum;
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least numNeurons kernels are launched.
// Multiply one encoder row by the sum vector for the corresponding ensemble. Then integrate to determine whether the neuron corresponding to that encoder row should spike. Results stored in spikes.
__global__ void encode(int totalNumNeurons, float* encoders, float* sums, float* encodeResult, int* encoderRowToEnsembleIndexor, int* ensembleOffsetInDimension, int* ensembleDimension, int* encoderStride, int* neuronIndexor)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < totalNumNeurons)
  {
    int ensembleIndex = encoderRowToEnsembleIndexor[i];
    int currentEnsembleDimension = ensembleDimension[ensembleIndex];
    int dimensionOffset = ensembleOffsetInDimension[ensembleIndex];

    int j, encoderOffset = i;
    float dot_product = 0;


    for(j = 0; j < currentEnsembleDimension; j++)
    {
      dot_product += encoders[encoderOffset] * sums[dimensionOffset + j];
      encoderOffset += encoderStride[j];
    }
    
    int neuronIndex = neuronIndexor[i];
    encodeResult[neuronIndex] = dot_product;
  }
}

__global__ void integrateAfterEncode(int numNeurons, float dt, float adjusted_dt, int steps, int* neuronToEnsembleIndexor, float* encodingResult, float* neuronVoltage, float* neuronReftime, float* tau_RC, float* tauRef, float* bias, float* scale, float* spikes, float* NDterminationSums, int* isSpikingEnsemble)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;
  
  if( i < numNeurons)
  {
    int ensembleIndex = neuronToEnsembleIndexor[i];
    float voltage = neuronVoltage[i];
    float refTime = neuronReftime[i];
    float tau_rc = tau_RC[ensembleIndex];
    float tau_ref = tauRef[ensembleIndex];
    float current = bias[i] + scale[i] * (encodingResult[i] + NDterminationSums[ensembleIndex]);

    if(isSpikingEnsemble[ensembleIndex])
    {
      float dV, post_ref, v_threshold = 1.0f;
      float spike_float;
      int j, spike = 0;

      for(j = 0; j < steps; j++)
      {
        dV = adjusted_dt / tau_rc * (current - voltage);
        voltage = max(voltage + dV, 0.0f);

        post_ref = 1.0f - (refTime - adjusted_dt) / adjusted_dt;

        voltage = post_ref >= 1.0f ? voltage : voltage * post_ref;

        voltage = post_ref <= 0.0f ? 0.0f : voltage;

        spike = spike ? spike : voltage > v_threshold;
        spike_float = spike ? 1.0f/dt : 0.0f;
        refTime = spike ? ((adjusted_dt / dV) * (dV - voltage + v_threshold)) + tau_ref : refTime - adjusted_dt;
        voltage = spike ? 0.0 : voltage;
      }

      neuronReftime[i] = refTime;
      neuronVoltage[i] = voltage;
      spikes[i] = spike_float;
    }
    else
    {
      spikes[i] = (current > 1.0) ? 1.0 / (tau_ref - tau_rc * logf(1.0 - 1.0 / current)) : 0.0;
    }
  }
}

// Kernel, run on GPU. block size and grid size should be set so that at least totalOutputSize kernels are launched.
// Multiply one decoder row by the spike vector for the corresponding ensemble. The result is one dimension of the output vector for the ensemble. Results stored in output.
__global__ void decode(int totalOutputSize, float* decoders, float* spikes, float* output, int* decoderRowToEnsembleIndexor, int* ensembleNumNeurons, int* ensembleOffsetInNeurons, int* decoderStride, int* outputIndexor)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;
  
  if( i < totalOutputSize)
  {
    
    int ensembleIndex = decoderRowToEnsembleIndexor[i];
    int numNeurons = ensembleNumNeurons[ensembleIndex];
    int spikesOffset = ensembleOffsetInNeurons[ensembleIndex];
    
    int j, decoderOffset = i;
    float dot_product = 0;

    for(j=0; j < numNeurons; j++)
    {
        dot_product += decoders[decoderOffset] * spikes[spikesOffset + j];

        decoderOffset += decoderStride[j];
    }
    

    int currentOutputIndex = outputIndexor[i];
    output[currentOutputIndex] = dot_product;
  }
}



// launch as many as there are ensembles
__global__ void processNDterminations(int numEnsembles, int numNDterminations, int steps, float adjusted_dt, int* NDterminationEnsembleOffset, int* terminationOffsetInInputs, int* terminationDimensions, int* inputIndex, float* input, float* weights, float* current, float* sum, float* tau)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < numEnsembles)
  {
    int offset = NDterminationEnsembleOffset[i];
    int count = (i == numEnsembles - 1) ? numNDterminations - offset : NDterminationEnsembleOffset[i+1] - offset;
    int j, k, terminationOffsetInInput, terminationDimension, index;
    float val, temp_sum = 0, temp_current, temp_tau;

    int weightIndexInEnsemble = i;

    if(count > 0)
    {
      for(j = 0; j < count; j++)
      {
        index = inputIndex[offset + j];
        terminationOffsetInInput = terminationOffsetInInputs[index]; 
        terminationDimension = terminationDimensions[index];

        val = 0;
        for(k = 0; k < terminationDimension; k++)
        {
          // have to figure out how to index this properly
          val += input[terminationOffsetInInput + k] * weights[weightIndexInEnsemble];
          weightIndexInEnsemble += numEnsembles;
        }

        temp_current = current[offset + j];
        temp_tau = tau[index];

        for(k = 0; k < steps; k++)
        {
          // testing this order, though this is the one used in the java code so it should work
          temp_current *= 1 - adjusted_dt / temp_tau;
          temp_current += val * adjusted_dt / temp_tau;
        }

        current[offset + j] = temp_current;
        
        temp_sum += temp_current;
      }

      sum[i] = temp_sum;
    }
  }
}


__global__ void moveGPUData(int size, int* map, float* to, float* from)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  if(i < size)
  {
    to[i] = from[ map[i] ];
  }
}
      
// run a NengoGPUData struct for one step
void run_NEFEnsembles(NengoGPUData* nengoData, float startTime, float endTime)
{
  float dt = endTime - startTime;

  //printf("start time: %f, end time %f, dt: %f, device: %d\n", startTime, endTime, dt, nengoData->device);

  hipError_t err;

  dim3 dimBlock(1, 1);
  dim3 dimGrid(1, 1);

//   int NDsteps = 
  //float NDadjusted_dt = dt / NDsteps; /// steps;
  int ND_steps = 1; //(int)(ceil(dt / nengoData->maxTimeStep));
  float ND_adjusted_dt = dt;// / ND_steps;

  int steps = 1;
  float adjusted_dt = dt;

//  if(((int) (startTime * 1000)) < 4)
  //printDynamicNengoGPUData(nengoData);


///////////////////////////////////////////////////////
// Copy input from host to GPU
///////////////////////////////////////////////////////

  hipMemcpy(nengoData->input->array + nengoData->GPUInputSize, sharedInput + nengoData->offsetInSharedInput, (nengoData->JavaInputSize + nengoData->CPUInputSize) * sizeof(float), hipMemcpyHostToDevice);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: copying cpu input to device");

///////////////////////////////////////////////////////
// Multiply input vectors by corresponding termination transform
///////////////////////////////////////////////////////
  dimBlock.x = 256;
  dimGrid.x = nengoData->totalNumTransformRows / dimBlock.x + 1;

  transform<<<dimGrid, dimBlock>>> (dt, nengoData->totalNumTransformRows, nengoData->input->array, nengoData->terminationOffsetInInput->array, nengoData->transformRowToInputIndexor->array, nengoData->terminationTransforms->array, nengoData->terminationTau->array, nengoData->terminationOutput->array, nengoData->terminationOutputIndexor->array, nengoData->inputDimension->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: transform");

///// sum the activation in each dimension of each ensemble

  dimBlock.x = 256;
  dimGrid.x = nengoData->totalEnsembleDimension / dimBlock.x + 1;

  sumTerminations <<<dimGrid, dimBlock>>> (nengoData->totalEnsembleDimension, nengoData->maxNumDecodedTerminations, nengoData->terminationOutput->array, nengoData->ensembleSums->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: sum");


///// process ND (nonDecoded) terminations
  dimBlock.x = 256;
  dimGrid.x = nengoData->numEnsembles / dimBlock.x + 1;

  processNDterminations<<<dimGrid, dimBlock>>>(nengoData->numEnsembles, nengoData->numNDterminations, ND_steps, ND_adjusted_dt, nengoData->NDterminationEnsembleOffset->array, nengoData->terminationOffsetInInput->array, nengoData->inputDimension->array, nengoData->NDterminationInputIndexor->array, nengoData->input->array, nengoData->NDterminationWeights->array, nengoData->NDterminationCurrents->array, nengoData->NDterminationEnsembleSums->array, nengoData->terminationTau->array);

  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: process non decoded");

///// encode
  dimBlock.x = 256;
  dimGrid.x = nengoData->numNeurons / dimBlock.x + 1;

  encode<<<dimGrid, dimBlock>>> (nengoData->numNeurons, nengoData->encoders->array, nengoData->ensembleSums->array, nengoData->encodeResult->array, nengoData->encoderRowToEnsembleIndexor->array, nengoData->ensembleOffsetInDimensions->array, nengoData->ensembleDimension->array, nengoData->encoderStride->array, nengoData->encoderRowToNeuronIndexor->array);


  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: encode");



///// integrate after encoding
  dimBlock.x = 256;
  dimGrid.x = nengoData->numNeurons / dimBlock.x + 1;

    integrateAfterEncode <<<dimGrid, dimBlock>>> (nengoData->numNeurons, dt, adjusted_dt, steps, nengoData->neuronToEnsembleIndexor->array, nengoData->encodeResult->array, nengoData->neuronVoltage->array, nengoData->neuronReftime->array, nengoData->ensembleTauRC->array, nengoData->ensembleTauRef->array, nengoData->neuronBias->array, nengoData->neuronScale->array, nengoData->spikes->array, nengoData->NDterminationEnsembleSums->array, nengoData->isSpikingEnsemble->array);

  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: integrate after encode");

///// decode

  dimBlock.x = 256;
  dimGrid.x = nengoData->totalOutputSize / dimBlock.x + 1;

  decode<<<dimGrid, dimBlock>>>(nengoData->totalOutputSize, nengoData->decoders->array, nengoData->spikes->array, nengoData->ensembleOutput->array, nengoData->decoderRowToEnsembleIndexor->array, nengoData->ensembleNumNeurons->array, nengoData->ensembleOffsetInNeurons->array, nengoData->decoderStride->array, nengoData->decoderRowToOutputIndexor->array);

  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: decode");


//// move output to device

  // reorganize the output, which comes out of decode in terms of ensembles, so that it is in terms of network arrays.
  dimGrid.x = nengoData->totalOutputSize / (dimBlock.x * dimBlock.y) + 1;
  moveGPUData<<<dimGrid, dimBlock>>>(nengoData->totalOutputSize, nengoData->ensembleOutputToNetworkArrayOutputMap->array, nengoData->output->array, nengoData->ensembleOutput->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: moveensembleoutput to network array output");

  if(nengoData->numSpikesToSendBack > 0)
  { 
    dimGrid.x = nengoData->numSpikesToSendBack / (dimBlock.x * dimBlock.y) + 1;
    moveGPUData<<<dimGrid, dimBlock>>>(nengoData->numSpikesToSendBack, nengoData->spikeMap->array, nengoData->output->array + nengoData->totalOutputSize, nengoData->spikes->array);
    err = hipGetLastError();
    checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: extract spikes to send back");
  }

  if(nengoData->CPUOutputSize + nengoData->numSpikesToSendBack > 0)
  {
    hipMemcpy(nengoData->outputHost->array, nengoData->output->array + nengoData->GPUOutputSize, (nengoData->CPUOutputSize + nengoData->numSpikesToSendBack) * sizeof(float), hipMemcpyDeviceToHost);
    err = hipGetLastError();
    checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: move output from GPU to CPU");
  }
  
//// move data along GPU projections
  dimGrid.x = nengoData->GPUInputSize / (dimBlock.x * dimBlock.y) + 1;
  moveGPUData<<<dimGrid, dimBlock>>>(nengoData->GPUInputSize, nengoData->GPUTerminationToOriginMap->array, nengoData->input->array, nengoData->output->array);
  err = hipGetLastError();
  checkCudaErrorWithDevice(err, nengoData->device, "run_NEFEnsembles: move output along GPU projections");
}

float* allocateCudaFloatArray(int size)
{
  float* temp;
  hipError_t err;
  err = hipMalloc((void**)&temp, size * sizeof(float));
  checkCudaError(err, "allocate cuda float array");
  return temp;
}
  
int* allocateCudaIntArray(int size)
{
  int* temp;
  hipError_t err;
  err = hipMalloc((void**)&temp, size * sizeof(int));
  checkCudaError(err, "allocate cuda int array");
  return temp;
}

long getDeviceCapacity(int device)
{
  hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);  
  return deviceProperties.totalGlobalMem;
}
  
void initializeDeviceInputAndOutput(NengoGPUData* nengoData)
{
  char* name;
  hipError_t err;

  name = "input";
  nengoData->input = newFloatArrayOnDevice(nengoData->totalInputSize, name); 
  
  name = "ensembleOutput";
  nengoData->ensembleOutput = newFloatArrayOnDevice(nengoData->totalOutputSize, name); 

  name = "output";
  nengoData->output = newFloatArrayOnDevice(nengoData->totalOutputSize + nengoData->numSpikesToSendBack, name); 
  
  name = "spikes";
  nengoData->spikes = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "terminationOutput";
  nengoData->terminationOutput = newFloatArrayOnDevice(nengoData->totalEnsembleDimension * nengoData->maxNumDecodedTerminations, name); 
  
  name = "ensembleSums";
  nengoData->ensembleSums = newFloatArrayOnDevice(nengoData->totalEnsembleDimension, name); 
  
  name = "encodeResult";
  nengoData->encodeResult = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "neuronVoltage";
  nengoData->neuronVoltage = newFloatArrayOnDevice(nengoData->numNeurons, name); 
  
  name = "neuronReftime";
  nengoData->neuronReftime = newFloatArrayOnDevice(nengoData->numNeurons, name); 


  err = hipMemset(nengoData->input->array, 0, nengoData->GPUInputSize * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->ensembleOutput->array, 0, nengoData->totalOutputSize * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->output->array, 0, (nengoData->totalOutputSize + nengoData->numSpikesToSendBack) * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->spikes->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->terminationOutput->array, 0, nengoData->totalEnsembleDimension * nengoData->maxNumDecodedTerminations * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->neuronVoltage->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->neuronReftime->array, 0, nengoData->numNeurons * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  
  name = "NDterminationCurrents";
  nengoData->NDterminationCurrents = newFloatArrayOnDevice(nengoData->numNDterminations, name); 
  name = "NDterminationEnsembleSum";
  nengoData->NDterminationEnsembleSums = newFloatArrayOnDevice(nengoData->numEnsembles, name); 

  err = hipMemset(nengoData->NDterminationCurrents->array, 0, nengoData->numNDterminations * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
  err = hipMemset(nengoData->NDterminationEnsembleSums->array, 0, nengoData->numEnsembles * sizeof(float));
  checkCudaErrorWithDevice(err, nengoData->device, "cuda setup structures");
}

#ifdef __cplusplus
}
#endif

