#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C"
{
#endif

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <cula_lapack_device.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "customCudaUtils.h"
#include "NengoUtilsGPU.h"

/*
  Inverts a square diagonal matrix in place. Launch n kernels where the 
  n is the length of one side of the matrix to be inverted.
*/
__global__ void invertS(float* Sd, int minDim, int leadingDim, float minSV, int numSV)
{
  int i = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;
  float valueInS;
  if(i < minDim)
  { 
     valueInS = Sd[i];
     Sd[i] = 0;

     if(valueInS < minSV || (numSV != -1 && i > numSV))
     {
          Sd[leadingDim*i + i] = 0;
     }
     else if(valueInS != 0)
     {
          Sd[leadingDim*i + i] = 1/valueInS;
     }
  }

  return;
}

/*
  Takes a square symmetrix matrix in lower triangular form (only the values below the main diagonal
  are present, the rest are probably NANs) and expands it to normal storage form.
  Launch n^2 kernels where n is the length of one side of the matrix. 
*/
__global__ void undoLowerTriangularStorage(float* Sd, unsigned int numElements, int stride)
{
  unsigned int index = threadIdx.x + (blockDim.x * threadIdx.y) + (blockIdx.x + (gridDim.x * blockIdx.y)) * blockDim.x * blockDim.y;

  int i = index / stride;
  int j = index % stride;

  if(i < j)
  {
    Sd[j + stride * i] = Sd[i + stride * j];
  }
}

/*
Takes in a matrix with dimension M x N stored in column major format and returns its pseudoInverse
*/
float* pseudoInverse(float* A, int M, int N, float minSV, int numSV, int inputOnDevice, int outputOnDevice)
{
    char jobu = 'S';
    char jobv = 'S';

    int minDim = min(M,N);
    int maxDim = max(M,N);
    
    float* Ad = NULL;
    float* Sd = NULL;
    float* VTd = NULL;
    float* Ud = NULL;

    hipError_t err;
    culaStatus status;

    if(inputOnDevice)
    {
      Ad = A;
    }else{
      err = hipMalloc((void**)&Ad, M*N*sizeof(float));
      checkCudaError(err);

      err = hipMemcpy(Ad, A, M*N*sizeof(float), hipMemcpyHostToDevice);
      checkCudaError(err);

      free(A);
    }

    err = hipMalloc((void**)&Sd, M*N*sizeof(float));
    checkCudaError(err);
    err = hipMemset((void*)Sd, 0, M*N*sizeof(float));
    checkCudaError(err);

    err = hipMalloc((void**)&Ud, minDim*M*sizeof(float));
    checkCudaError(err);

    err = hipMalloc((void**)&VTd, minDim*N*sizeof(float));
    checkCudaError(err);

    status = culaInitialize();
    checkStatus(status);

    status = culaDeviceSgesvd(jobu, jobv, M, N, Ad, M, Sd, Ud, M, VTd, minDim);  
    checkStatus(status);
   
   
    // we need at least minDim blocks...its ok to have more, but they will not be used
    dim3 dimBlock(16, 16);
    dim3 dimGrid(1, minDim / (dimBlock.x * dimBlock.y) + 1);
    // Now U should be in Ad, S in Sd and VT in VTd
    invertS<<<dimGrid,dimBlock>>>(Sd, minDim, minDim, minSV, numSV);
    checkCudaError(err);


    cublasInit();
   
    // compute S^-1 * UT and store in Ad
    hipblasSgemm('N', 'T', minDim, M, minDim, 1, Sd, minDim, Ud, M, 0, Ad, minDim);

    // compute V * S^-1 * UT and store in Sd
    hipblasSgemm('T', 'N', N, M, minDim, 1, VTd, minDim, Ad, minDim, 0, Sd, N);
    
    //Shut everything down
    cublasShutdown();

    err = hipFree(Ad);
    checkCudaError(err);

    err = hipFree(VTd);
    checkCudaError(err);

    err = hipFree(Ud);
    checkCudaError(err);

    culaFreeBuffers();

    if(outputOnDevice)
    {
      return Sd;
    }else{
      float* S = (float*) malloc(M*N*sizeof(float));

      err = hipMemcpy(S, Sd, M*N*sizeof(float), hipMemcpyDeviceToHost);
      checkCudaError(err);

      hipFree(Sd);

      return S;
    }
}



/*
Takes in A_transpose stored in column major format and returns A * A_transpose
*/
float* findGamma(float* A_transpose, int numNeurons, int numEvalPoints, int inputOnDevice, int outputOnDevice)
{
    hipError_t err;

    char uplo = 'U';
    char trans = 'T';

    float alpha = 1.0f / ((float) numEvalPoints);

    float* A_transpose_d;

    if(inputOnDevice)
    {
      A_transpose_d = A_transpose;
    }else{
      err = hipMalloc((void**)&A_transpose_d, numNeurons*numEvalPoints*sizeof(float));
      checkCudaError(err);

      err = hipMemcpy(A_transpose_d, A_transpose, numNeurons*numEvalPoints*sizeof(float), hipMemcpyHostToDevice);
      checkCudaError(err);

      free(A_transpose);
    }

    float* gamma_d = NULL;
    err = hipMalloc((void**)&gamma_d, numNeurons*numNeurons*sizeof(float));
    checkCudaError(err);

    cublasInit();

    // A times A transpose stored in matrix gamma_d
    hipblasSsyrk(uplo, trans, numNeurons, numEvalPoints, alpha, A_transpose_d, numEvalPoints, 0, gamma_d, numNeurons);

    hipFree(A_transpose_d);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(numNeurons / dimBlock.x + 1, numNeurons / dimBlock.y + 1);

    undoLowerTriangularStorage<<<dimGrid, dimBlock>>>(gamma_d, numNeurons * numNeurons, numNeurons);
    err = hipGetLastError();
    checkCudaError(err);

    if(outputOnDevice)
    {
      return gamma_d;
    }else{
      float* gamma = (float*) malloc(numNeurons * numNeurons * sizeof(float));

      err = hipMemcpy(gamma, gamma_d, numNeurons*numNeurons*sizeof(float), hipMemcpyDeviceToHost);
      checkCudaError(err);
      
      err = hipFree(gamma_d);
      checkCudaError(err);

      return gamma;
    }
}

#ifdef __cplusplus
}
#endif
